

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

extern "C"
{

bool cuda_available()
{
    int device_count = 0;
    hipError_t err = hipGetDeviceCount(&device_count);
    if (err != hipSuccess) {
        printf("CUDA Error in cuda_available: %s\n", hipGetErrorString(err));
        return false;
    }
    printf("Found %d CUDA device(s)\n", device_count);
    return device_count > 0;
}

void cuda_check(hipError_t result, const char *msg)
{
    if (result != hipSuccess)
    {
        printf("CUDA Error: %s - %s\n", msg, hipGetErrorString(result));
        fflush(stdout);
        exit(EXIT_FAILURE);
    }
}

void *cuda_malloc(size_t size)
{
    if (size == 0) {
        printf("Warning: Attempting to allocate 0 bytes\n");
        return nullptr;
    }
    
    void *new_ptr = nullptr;
    hipError_t err = hipMalloc(&new_ptr, size);
    cuda_check(err, "hipMalloc in cuda_malloc");
    
    printf("Allocated %zu bytes at GPU address %p\n", size, new_ptr);
    return new_ptr;
}

void cuda_free(void *ptr)
{
    if (ptr != nullptr) {
        hipError_t err = hipFree(ptr);
        if (err != hipSuccess) {
            printf("CUDA Error in cuda_free: %s\n", hipGetErrorString(err));
        }
    }
}

void cuda_sync()
{
    cuda_check(hipDeviceSynchronize(), "hipDeviceSynchronize");
}

void cuda_copy_to_gpu(void *dest_ptr, const void *src_ptr, size_t count)
{
    if (dest_ptr == nullptr) {
        printf("Error: dest_ptr is null in cuda_copy_to_gpu\n");
        exit(EXIT_FAILURE);
    }
    if (src_ptr == nullptr) {
        printf("Error: src_ptr is null in cuda_copy_to_gpu\n");
        exit(EXIT_FAILURE);
    }
    cuda_check(hipMemcpy(dest_ptr, src_ptr, count, hipMemcpyHostToDevice), "cuda_copy_to_gpu");
}

void cuda_copy_to_cpu(void *dest_ptr, const void *src_ptr, size_t count)
{
    if (dest_ptr == nullptr) {
        printf("Error: dest_ptr is null in cuda_copy_to_cpu\n");
        exit(EXIT_FAILURE);
    }
    if (src_ptr == nullptr) {
        printf("Error: src_ptr is null in cuda_copy_to_cpu\n");
        exit(EXIT_FAILURE);
    }
    cuda_check(hipMemcpy(dest_ptr, src_ptr, count, hipMemcpyDeviceToHost), "cuda_copy_to_cpu");
}

__global__ void cuda_add_float_kernel(const float *a, const float *b, float *c, size_t n)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        c[idx] = a[idx] + b[idx];
    }
}

void cuda_add_elementwise(const float *a_host, const float *b_host, float *c_host, size_t n)
{
    // Input validation
    if (a_host == nullptr || b_host == nullptr || c_host == nullptr) {
        printf("Error: null pointer passed to cuda_add_elementwise\n");
        return;
    }
    
    if (n == 0) {
        printf("Warning: n=0 in cuda_add_elementwise\n");
        return;
    }
    
    printf("Starting cuda_add_elementwise with n=%zu\n", n);
    
    // Check if CUDA is available
    if (!cuda_available()) {
        printf("Error: CUDA not available\n");
        return;
    }
    
    float *d_a = nullptr;
    float *d_b = nullptr;
    float *d_c = nullptr;
    
    size_t bytes = n * sizeof(float);
    printf("Allocating %zu bytes per array\n", bytes);
    
    // Allocate device memory
    cuda_check(hipMalloc(&d_a, bytes), "hipMalloc d_a");
    cuda_check(hipMalloc(&d_b, bytes), "hipMalloc d_b");
    cuda_check(hipMalloc(&d_c, bytes), "hipMalloc d_c");
    
    printf("Device memory allocated successfully\n");
    
    // Copy data to device
    cuda_check(hipMemcpy(d_a, a_host, bytes, hipMemcpyHostToDevice), "hipMemcpy d_a");
    cuda_check(hipMemcpy(d_b, b_host, bytes, hipMemcpyHostToDevice), "hipMemcpy d_b");
    
    printf("Data copied to device\n");
    
    // Launch kernel
    const int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    
    printf("Launching kernel with grid_size=%d, block_size=%d\n", grid_size, block_size);
    
    cuda_add_float_kernel<<<grid_size, block_size>>>(d_a, d_b, d_c, n);
    
    // Check for kernel launch errors
    hipError_t kernel_err = hipGetLastError();
    cuda_check(kernel_err, "cuda kernel launch");
    
    // Synchronize
    cuda_check(hipDeviceSynchronize(), "cuda kernel sync");
    
    printf("Kernel executed successfully\n");
    
    // Copy result back
    cuda_check(hipMemcpy(c_host, d_c, bytes, hipMemcpyDeviceToHost), "hipMemcpy c_host");
    
    printf("Result copied back to host\n");
    
    // Free device memory
    cuda_check(hipFree(d_a), "hipFree d_a");
    cuda_check(hipFree(d_b), "hipFree d_b");
    cuda_check(hipFree(d_c), "hipFree d_c");
    
    printf("Device memory freed\n");
}
}