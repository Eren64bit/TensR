#include <tensr_cuda_api.cuh>
#include <stdio.h>

bool cuda_available()
{
    int device_count = 0;
    hipError_t err = hipGetDeviceCount(&device_count);
    return (err == hipSuccess) && (device_count > 0);
}

void cuda_check(int result, const char *msg = "CUDA error")
{
    hipError_t cuda_result = static_cast<hipError_t>(result);
    if (cuda_result != hipSuccess)
    {
        printf("CUDA Error: %s - %s\n", msg, hipGetErrorString(cuda_result));
    } 
}

void *cuda_malloc(size_t size)
{
    void *new_ptr = nullptr;
    cuda_check(hipMalloc(&new_ptr, size));
    return new_ptr;
}

void cuda_free(void *ptr)
{
    hipFree(ptr);
}

void cuda_sync()
{
    hipDeviceSynchronize();
}

void cuda_copy_to_gpu(void *dest_ptr, const void *src_ptr, size_t count)
{
    cuda_check(hipMemcpy(dest_ptr, src_ptr, count, hipMemcpyHostToDevice));
}

void cuda_copy_to_cpu(void *dest_ptr, const void *src_ptr, size_t count)
{
    cuda_check(hipMemcpy(dest_ptr, src_ptr, count, hipMemcpyDeviceToHost));
}